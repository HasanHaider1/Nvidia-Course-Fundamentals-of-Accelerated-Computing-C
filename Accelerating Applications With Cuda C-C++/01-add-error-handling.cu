
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * Add error handling to this source code to learn what errors
   * exist, and then correct them. Googling error messages may be
   * of service if actions for resolving them are not clear to you.
   */
   

  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  
  
  hipMallocManaged(&a, size);
  
  
  init(a, N);

  size_t threads_per_block = 1024;
  size_t number_of_blocks = 32;

  hipError_t Error1, Error2;
  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  
  Error1 = hipGetLastError();
  Error2 = hipDeviceSynchronize();
 
  if (Error1 != hipSuccess) printf("Error: %s\n", hipGetErrorString(Error1));
  if (Error2 != hipSuccess) printf("Error: %s\n", hipGetErrorString(Error2));
  
  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
